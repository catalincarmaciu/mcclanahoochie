#include "hip/hip_runtime.h"
/*
 * cpu_autocorrelation_kernel.cu
 *
 *  Created on: Feb 23, 2010
 *      Author: chris
 */

/////////////////////////////////////
// imports
/////////////////////////////////////
#include "cpu_common.cuh"

#include <thrust/host_vector.h>
#include <thrust/transform_reduce.h>
#include <thrust/extrema.h>

/////////////////////////////////////
// data extraction kernel
/////////////////////////////////////
__global__ void cpu_extract_xyz_kernel(int N, void* outAx, void* outAy, void* outAz) {

}

/////////////////////////////////////
// external data extraction kernel manager
/////////////////////////////////////
void cpu_extract_xyz(float* h_xyz, const int validBodies, float* h_x, float* h_y, float* h_z) {

    for (int i = 0; i < validBodies; ++i) {
        h_x[i] = h_xyz[i * 3 + 0];
        h_y[i] = h_xyz[i * 3 + 1];
        h_z[i] = h_xyz[i * 3 + 2];
    }

}

/////////////////////////////////////
// core coviariance vector calculations (float)
/////////////////////////////////////
struct cov_functor_f1 {
    const float u1, u2;

    cov_functor_f1(float _u1, float _u2) : u1(_u1), u2(_u2) {}

    __host__ __host__
    float operator()(const float& t1, const float& t2) const {
        return (t1 - u1) * (t2 - u2);
    }
};

struct cov_functor_f2 {
    cov_functor_f2() {}

    __host__ __host__
    float operator()(const float& t1, const float& t2) const {
        return ((t1 - t2) * (t1 - t2));
    }
};

/////////////////////////////////////
// external compute autocorrelation between datasets at time1 and time2 (float)
/////////////////////////////////////
float cpu_compute_autocorrelation(thrust::host_vector<float>& data_t1, thrust::host_vector<float>& data_t2, int N, int type) {
    // temp
    thrust::host_vector<float> data_r(N);
    float ac = 0.0f;

    switch (type) {
    case 1: {
        // http://en.wikipedia.org/wiki/Pearson_product-moment_correlation_coefficient

        // find means
        float u1 = thrust::reduce(data_t1.begin(), data_t1.end(), (float)0, thrust::plus<float>()) / N;
        float u2 = thrust::reduce(data_t2.begin(), data_t2.end(), (float)0, thrust::plus<float>()) / N;

        // r = (t1-u1)*(t2-u2)
        thrust::transform(data_t1.begin(), data_t1.end(), data_t2.begin(), data_r.begin(), cov_functor_f1(u1, u2));

        // cov = sum(r_vector) / n
        float cov = thrust::reduce(data_r.begin(), data_r.end(), (float)0, thrust::plus<float>()) / N;

        // variances
        thrust::transform(data_t1.begin(), data_t1.end(), data_t1.begin(), data_r.begin(), cov_functor_f1(u1, u1));
        float var1 = thrust::reduce(data_r.begin(), data_r.end(), (float)0, thrust::plus<float>()) / N;
        thrust::transform(data_t2.begin(), data_t2.end(), data_t2.begin(), data_r.begin(), cov_functor_f1(u2, u2));
        float var2 = thrust::reduce(data_r.begin(), data_r.end(), (float)0, thrust::plus<float>()) / N;

        // standard deviations
        float std1 = sqrt(var1);
        float std2 = sqrt(var2);

        // autocorrelation
        ac = cov / (std1 * std2);
        break;
    }
    case 2: {
        // http://en.wikipedia.org/wiki/Durbin-Watson_statistic

        // r = (t1-t2)^2
        thrust::transform(data_t1.begin(), data_t1.end(), data_t2.begin(), data_r.begin(), cov_functor_f2());
        float et2 = thrust::reduce(data_r.begin(), data_r.end(), (float)0, thrust::plus<float>()) ;

        // r = t1^2
        thrust::transform(data_t1.begin(), data_t1.end(), data_t1.begin(), data_r.begin(), cov_functor_f1(0, 0));
        float et1 = thrust::reduce(data_r.begin(), data_r.end(), (float)0, thrust::plus<float>()) ;

        // autocorrelation
        ac = 1 - et2 / et1;
        break;
    }

    } // end case

    // out
    return ac;
}

/////////////////////////////////////
// core coviariance vector calculations (int)
/////////////////////////////////////
struct cov_functor_i1 {
    const float u1, u2;

    cov_functor_i1(float _u1, float _u2) : u1(_u1), u2(_u2) {}

    __host__ __host__
    float operator()(const int& t1, const int& t2) const {
        return (t1 - u1) * (t2 - u2);
    }
};

struct cov_functor_i2 {
    cov_functor_i2() {}

    __host__ __host__
    float operator()(const int& t1, const int& t2) const {
        return ((t1 - t2) * (t1 - t2));
    }
};

/////////////////////////////////////
// external compute autocorrelation between datasets at time1 and time2 (int)
/////////////////////////////////////
float cpu_compute_autocorrelation(thrust::host_vector<int>& data_t1, thrust::host_vector<int>& data_t2, int N, int type) {
    // temp
    thrust::host_vector<float> data_r(N);
    float ac = 0.0f;

    switch (type) {
    case 1: {
        // http://en.wikipedia.org/wiki/Pearson_product-moment_correlation_coefficient

        // find means
        float u1 = thrust::reduce(data_t1.begin(), data_t1.end(), (int)0, thrust::plus<int>()) / N;
        float u2 = thrust::reduce(data_t2.begin(), data_t2.end(), (int)0, thrust::plus<int>()) / N;

        // r = (t1-u1)*(t2-u2)
        thrust::transform(data_t1.begin(), data_t1.end(), data_t2.begin(), data_r.begin(), cov_functor_i1(u1, u2));

        // cov = sum(r_vector) / n
        float cov = thrust::reduce(data_r.begin(), data_r.end(), (float)0, thrust::plus<float>()) / N;

        // variances
        thrust::transform(data_t1.begin(), data_t1.end(), data_t1.begin(), data_r.begin(), cov_functor_f1(u1, u1));
        float var1 = thrust::reduce(data_r.begin(), data_r.end(), (float)0, thrust::plus<float>()) / N;
        thrust::transform(data_t2.begin(), data_t2.end(), data_t2.begin(), data_r.begin(), cov_functor_f1(u2, u2));
        float var2 = thrust::reduce(data_r.begin(), data_r.end(), (float)0, thrust::plus<float>()) / N;

        // standard deviations
        float std1 = sqrt(var1);
        float std2 = sqrt(var2);

        // autocorrelation
        ac = cov / (std1 * std2);
        break;
    }
    case 2: {
        // http://en.wikipedia.org/wiki/Durbin-Watson_statistic

        // r = (t1-t2)^2
        thrust::transform(data_t1.begin(), data_t1.end(), data_t2.begin(), data_r.begin(), cov_functor_i2());
        float et2 = thrust::reduce(data_r.begin(), data_r.end(), (float)0, thrust::plus<float>()) ;

        // r = t1^2
        thrust::transform(data_t1.begin(), data_t1.end(), data_t1.begin(), data_r.begin(), cov_functor_f1(0, 0));
        float et1 = thrust::reduce(data_r.begin(), data_r.end(), (float)0, thrust::plus<float>()) ;

        // autocorrelation
        ac = 1 - et2 / et1;
        break;
    }

    } // end case

    // out
    return ac;
}

