#include "hip/hip_runtime.h"
// compile:   nvcc -O3  foo.cu -lcublas -I/usr/local/cuda/include -L/usr/local/cuda/lib64 */
// run: ./test_sgemm 128 256   # run for that range

#include <stdio.h>
#include <hip/hip_runtime.h>
#include <hipblas.h>
#include <err.h>

#define REPS  7

int main(int argc, char* args[]) {
    if (argc != 3) { errx(-1, "%s <small> <large>", args[0]); }

    int small = atoi(args[1]);
    int large = atoi(args[2]);


    int runtimeVersion;
    hipRuntimeGetVersion(&runtimeVersion);
    printf("toolkit version %d\n", runtimeVersion);

    float* d_A, *d_B, *d_C;
    int bytes = large * large * sizeof(float);
    hipMalloc((void**)&d_A, bytes);
    hipMalloc((void**)&d_B, bytes);
    hipMalloc((void**)&d_C, bytes);

    /* prepare timers */
    hipEvent_t start, stop;
    hipEventCreate(&start);
    hipEventCreate(&stop);

    hipblasSgemm('N', 'N', small, small, small, 1, d_A, small, d_B, small, 0, d_C, small);
    hipDeviceSynchronize();

    for (int n = small; n <= large; ++n) {
        hipEventRecord(start, 0);

        for (int reps = 0; reps < REPS; ++reps)
        { hipblasSgemm('N', 'N', n, n, n, 1, d_A, n, d_B, n, 0, d_C, n); }

        hipEventRecord(stop, 0);
        hipEventSynchronize(stop);
        float time;
        hipEventElapsedTime(&time, start, stop);
        double gflops = pow(n, 3) * 2.0 * REPS * 1e-6 / time;
        printf("%4d  %g\n", n, gflops);
    }

    return 0;
}
